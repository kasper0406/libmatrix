
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#define BLOCK_SIZE 128

#ifdef DEBUG
#define DEBUG_SYNCHRONIZE() cudaDeviceSynchronize()
#else
#define DEBUG_SYNCHRONIZE()
#endif

__global__ void add(int N, float* A, float* B, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        R[i] = A[i] + B[i];
    }
}

__global__ void add_assign(int N, float* A, float* B) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        A[i] += B[i];
    }
}

__global__ void sub(int N, float* A, float* B, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        R[i] = A[i] - B[i];
    }
}

__global__ void entrywise_multiply(int N, float* A, float* B, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        R[i] = A[i] * B[i];
    }
}

__global__ void scalar_multiply(int N, float scalar, float* A, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        R[i] = scalar * A[i];
    }
}

// TODO(knielsen): Make this more efficient
__global__ void multiply(int A_rows, int A_cols, float* A,
                         int B_rows, int B_cols, float* B,
                         int N, float* R) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < A_rows && col < B_cols) {
        int scalar = 0;
        for (int k = 0; k < A_cols; k++) {
            scalar += A[row * A_cols + k] * B[k * B_cols + col];
        }
        R[row * B_cols + col] = scalar;
    }
}

// TODO(knielsen): Make this more efficient
__global__ void transpose(int rows, int columns, float* input, float* R) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < rows && col < columns) {
        R[col * rows + row] = input[row * columns + col];
    }
}

__global__ void add_constant_row(float padding, int N, int columns, float* A, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        R[i] = i >= columns ? A[i - columns] : padding;
    }
}

__global__ void setup_random_state(hiprandState *state, int N, size_t seed) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        hiprand_init(seed, index, 0, &state[index]);
    }
}

__global__ void dropout_elements(hiprandState *state, float rate, int N, float* A, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState local_state = state[index];
    for (int i = index; i < N; i += stride) {
        if (hiprand_uniform(&local_state) < rate) {
            R[i] = 0;
        } else {
            R[i] = A[i];
        }
    }
    state[index] = local_state;
}

__global__ void dropout_rows(float rate, int columns, float* A, float* R, size_t seed) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(seed, row, 0, &state);
    float discriminator = hiprand_uniform(&state);

    for (int i = 0; i < columns; i++) {
        R[row * columns + i] = (discriminator < rate) ? 0 : A[row * columns + i];
    }
}

__global__ void copy(int N, float* A, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        R[i] = A[i];
    }
}

__global__ void apply_sigmoid(int N, float* A, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        float exponential = exp(A[i]);
        R[i] = exponential / (exponential + 1);
    }
}

__global__ void apply_sigmoid_derivative(int N, float* A, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        float exponential = exp(A[i]);
        R[i] = exponential / (1 + exponential * (exponential + 2));
    }
}

__global__ void apply_relu(int N, float* A, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        R[i] = A[i] < 0 ? 0 : A[i];
    }
}

__global__ void apply_relu_derivative(int N, float* A, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        R[i] = A[i] < 0 ? 0 : 1;
    }
}

__global__ void apply_twoplayerscore(int N, float* A, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        float exponential = exp(A[i]);
        R[i] = (exponential - 1) / (exponential + 1);
    }
}

__global__ void apply_twoplayerscore_derivative(int N, float* A, float* R) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < N; i += stride) {
        float exponential = exp(A[i]);
        R[i] = (2 * exponential) / (1 + exponential * (exponential + 2));
    }
}

extern "C" {

    struct MatrixHandle {
        size_t rows;
        size_t columns;
        float* elements;
    };

    void matrix_synchronize(bool only_current_thread) {
        if (only_current_thread) {
            hipStreamSynchronize(hipStreamPerThread);
        } else {
            hipDeviceSynchronize();
        }
    }

    int matrix_alloc(size_t rows, size_t columns, float* elements, struct MatrixHandle* handle) {
        const size_t N = rows * columns;

        handle->rows = rows;
        handle->columns = columns;
        auto alloc_res = hipMallocManaged(&handle->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        for (size_t i = 0; i < N; i++) {
            handle->elements[i] = elements[i];
        }

        return 0;
    }

    void matrix_free(MatrixHandle* handle) {
        hipFree(handle->elements);
    }

    int matrix_add(const MatrixHandle* A, const MatrixHandle* B, MatrixHandle* result_handle) {
        if (A->rows != B->rows || A->columns != B->columns) {
            return 20;
        }

        const auto N = A->rows * A->columns;

        result_handle->rows = A->rows;
        result_handle->columns = A->columns;
        auto alloc_res = hipMallocManaged(&result_handle->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int blockSize = 128;
        int numBlocks = (N + blockSize - 1) / blockSize;
        add<<<numBlocks, blockSize>>>(N, A->elements, B->elements, result_handle->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_add_assign(MatrixHandle* A, const MatrixHandle* B) {
        if (A->rows != B->rows || A->columns != B->columns) {
            return 20;
        }

        const auto N = A->rows * A->columns;
        int blockSize = 128;
        int numBlocks = (N + blockSize - 1) / blockSize;
        add_assign<<<numBlocks, blockSize>>>(N, A->elements, B->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_sub(const MatrixHandle* A, const MatrixHandle* B, MatrixHandle* result_handle) {
        if (A->rows != B->rows || A->columns != B->columns) {
            return 20;
        }

        const auto N = A->rows * A->columns;

        result_handle->rows = A->rows;
        result_handle->columns = A->columns;
        auto alloc_res = hipMallocManaged(&result_handle->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int blockSize = 128;
        int numBlocks = (N + blockSize - 1) / blockSize;
        sub<<<numBlocks, blockSize>>>(N, A->elements, B->elements, result_handle->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_entrywise_multiply(const MatrixHandle* A, const MatrixHandle* B, MatrixHandle* result_handle) {
        if (A->rows != B->rows || A->columns != B->columns) {
            return 20;
        }

        const auto N = A->rows * A->columns;

        result_handle->rows = A->rows;
        result_handle->columns = A->columns;
        auto alloc_res = hipMallocManaged(&result_handle->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int blockSize = 128;
        int numBlocks = (N + blockSize - 1) / blockSize;
        entrywise_multiply<<<numBlocks, blockSize>>>(N, A->elements, B->elements, result_handle->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_scalar_multiply(const MatrixHandle* A, float scalar, MatrixHandle* result_handle) {
        const auto N = A->rows * A->columns;
        result_handle->rows = A->rows;
        result_handle->columns = A->columns;
        auto alloc_res = hipMallocManaged(&result_handle->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int blockSize = 128;
        int numBlocks = (N + blockSize - 1) / blockSize;
        scalar_multiply<<<numBlocks, blockSize>>>(N, scalar, A->elements, result_handle->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_multiply(const MatrixHandle* A, const MatrixHandle* B, MatrixHandle* result_handle) {
        if (A->columns != B->rows) {
            return 30;
        }

        const auto N_result = A->rows * B->columns;

        result_handle->rows = A->rows;
        result_handle->columns = B->columns;
        auto alloc_res = hipMallocManaged(&result_handle->elements, N_result * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        const int block_size = 16;
        dim3 threads_per_block(block_size, block_size);
        dim3 num_blocks((A->rows + block_size - 1) / block_size,
                        (B->columns + block_size - 1) / block_size);
        multiply<<<num_blocks, threads_per_block>>>(A->rows, A->columns, A->elements,
                                                    B->rows, B->columns, B->elements,
                                                    N_result, result_handle->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_transpose(const MatrixHandle* A, MatrixHandle* result) {
        const auto N = A->rows * A->columns;

        result->rows = A->columns;
        result->columns = A->rows;
        auto alloc_res = hipMallocManaged(&result->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        const int block_size = 16;
        dim3 threads_per_block(block_size, block_size);
        dim3 num_blocks((A->rows + block_size - 1) / block_size,
                        (A->columns + block_size - 1) / block_size);
        transpose<<<num_blocks, threads_per_block>>>(A->rows, A->columns, A->elements, result->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_add_constant_row(float padding, const MatrixHandle* A, MatrixHandle* result) {
        result->rows = A->rows + 1;
        result->columns = A->columns;
        const auto N = result->rows * result->columns;
        auto alloc_res = hipMallocManaged(&result->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int blockSize = 128;
        int numBlocks = (N + blockSize - 1) / blockSize;
        add_constant_row<<<numBlocks, blockSize>>>(padding, N, A->columns, A->elements, result->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_dropout_elements(float rate, const MatrixHandle* A, MatrixHandle* result) {
        result->rows = A->rows;
        result->columns = A->columns;
        const auto N = result->rows * result->columns;
        auto alloc_res = hipMallocManaged(&result->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        hiprandState* random_state;
        alloc_res = hipMalloc(&random_state, N * sizeof(hiprandState));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        srand(time(NULL));

        int blockSize = BLOCK_SIZE;
        int numBlocks = (N + blockSize - 1) / blockSize;
        setup_random_state<<<numBlocks, blockSize>>>(random_state, N, rand());
        dropout_elements<<<numBlocks, blockSize>>>(random_state, rate, N, A->elements, result->elements);

        DEBUG_SYNCHRONIZE();
        hipFree(random_state);

        return 0;
    }

    int matrix_dropout_rows(float rate, const MatrixHandle* A, MatrixHandle* result) {
        result->rows = A->rows;
        result->columns = A->columns;
        const auto N = result->rows * result->columns;
        auto alloc_res = hipMallocManaged(&result->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        srand(time(NULL));
        size_t seed = rand();
        dim3 threads_per_block(1, A->columns);
        dim3 num_blocks(A->rows, 1);
        dropout_rows<<<num_blocks, threads_per_block>>>(rate, A->columns, A->elements, result->elements, seed);

        DEBUG_SYNCHRONIZE();

        return 0;
    }
    
    int matrix_copy(const MatrixHandle* A, MatrixHandle* result) {
        result->rows = A->rows;
        result->columns = A->columns;
        const auto N = result->rows * result->columns;
        auto alloc_res = hipMallocManaged(&result->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
        copy<<<num_blocks, BLOCK_SIZE>>>(N, A->elements, result->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_apply_sigmoid(const MatrixHandle* A, MatrixHandle* result) {
        result->rows = A->rows;
        result->columns = A->columns;
        const auto N = result->rows * result->columns;
        auto alloc_res = hipMallocManaged(&result->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
        apply_sigmoid<<<num_blocks, BLOCK_SIZE>>>(N, A->elements, result->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_apply_sigmoid_derivative(const MatrixHandle* A, MatrixHandle* result) {
        result->rows = A->rows;
        result->columns = A->columns;
        const auto N = result->rows * result->columns;
        auto alloc_res = hipMallocManaged(&result->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
        apply_sigmoid_derivative<<<num_blocks, BLOCK_SIZE>>>(N, A->elements, result->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_apply_relu(const MatrixHandle* A, MatrixHandle* result) {
        result->rows = A->rows;
        result->columns = A->columns;
        const auto N = result->rows * result->columns;
        auto alloc_res = hipMallocManaged(&result->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
        apply_relu<<<num_blocks, BLOCK_SIZE>>>(N, A->elements, result->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_apply_relu_derivative(const MatrixHandle* A, MatrixHandle* result) {
        result->rows = A->rows;
        result->columns = A->columns;
        const auto N = result->rows * result->columns;
        auto alloc_res = hipMallocManaged(&result->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
        apply_relu_derivative<<<num_blocks, BLOCK_SIZE>>>(N, A->elements, result->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_apply_twoplayerscore(const MatrixHandle* A, MatrixHandle* result) {
        result->rows = A->rows;
        result->columns = A->columns;
        const auto N = result->rows * result->columns;
        auto alloc_res = hipMallocManaged(&result->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
        apply_twoplayerscore<<<num_blocks, BLOCK_SIZE>>>(N, A->elements, result->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }

    int matrix_apply_twoplayerscore_derivative(const MatrixHandle* A, MatrixHandle* result) {
        result->rows = A->rows;
        result->columns = A->columns;
        const auto N = result->rows * result->columns;
        auto alloc_res = hipMallocManaged(&result->elements, N * sizeof(float));
        if (alloc_res != hipSuccess) {
            return 10;
        }

        int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
        apply_twoplayerscore_derivative<<<num_blocks, BLOCK_SIZE>>>(N, A->elements, result->elements);

        DEBUG_SYNCHRONIZE();

        return 0;
    }
}
